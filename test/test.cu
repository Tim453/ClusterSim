#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "matmul.cuh"





TEST(MatrtixMultiply, NonCluster) {
    int MatSize = 64;
    int Tile_size = 8;
    int smem_size = Tile_size * Tile_size * sizeof(int) * 2;
    int *input_a, *input_b, *output, *gpu_output;
    input_a = new int[MatSize * MatSize];
    input_b = new int[MatSize * MatSize];
    output = new int[MatSize * MatSize];
    gpu_output = new int[MatSize * MatSize];

    init_mat(input_a, MatSize);
    init_mat(input_b, MatSize);
    matmul_cpu(input_a, input_b, output, MatSize);

    int *d_input_a, *d_input_b, *d_output;
    hipMalloc(&d_input_a, MatSize * MatSize * sizeof(int));
    hipMalloc(&d_input_b, MatSize * MatSize * sizeof(int));
    hipMalloc(&d_output, MatSize * MatSize * sizeof(int));
    hipMemcpy(d_input_a, input_a, MatSize * MatSize * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(d_input_b, input_b, MatSize * MatSize * sizeof(int),
               hipMemcpyHostToDevice);

    dim3 num_blocks(ceil(MatSize / (float)Tile_size),
                    ceil(MatSize / (float)Tile_size), 1);

    dim3 num_threads(Tile_size, Tile_size, 1);

    matmul_gpu<<<num_blocks, num_threads, smem_size>>>(
        d_input_a, d_input_b, d_output, MatSize, Tile_size);
    hipMemcpy(gpu_output, d_output, MatSize * MatSize * sizeof(int),
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for (int i = 0; i < MatSize; i++) {
        for (int j = 0; j < MatSize; j++) {
            int index = i * MatSize + j;
            EXPECT_EQ(output[index], gpu_output[index]);
        }
    }
}

TEST(MatrtixMultiply, Cluster) {
    const int MatSize = 64;
    const int Tile_size = 8;
    const int ClusterSize = 4;
    const int smem_size = Tile_size * Tile_size * sizeof(int) * 2;
    int *input_a, *input_b, *output, *gpu_output;
    input_a = new int[MatSize * MatSize];
    input_b = new int[MatSize * MatSize];
    output = new int[MatSize * MatSize];
    gpu_output = new int[MatSize * MatSize];

    init_mat(input_a, MatSize);
    init_mat(input_b, MatSize);
    matmul_cpu(input_a, input_b, output, MatSize);

    int *d_input_a, *d_input_b, *d_output;
    hipMalloc(&d_input_a, MatSize * MatSize * sizeof(int));
    hipMalloc(&d_input_b, MatSize * MatSize * sizeof(int));
    hipMalloc(&d_output, MatSize * MatSize * sizeof(int));
    hipMemcpy(d_input_a, input_a, MatSize * MatSize * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(d_input_b, input_b, MatSize * MatSize * sizeof(int),
               hipMemcpyHostToDevice);

    dim3 num_blocks(ceil(MatSize / (float)Tile_size),
                    ceil(MatSize / (float)Tile_size), 1);

    dim3 num_threads(Tile_size, Tile_size, 1);

    // Launch the kernel
    hipLaunchConfig_t config = {0};
    config.gridDim = num_blocks;
    config.blockDim = num_threads;
    config.dynamicSmemBytes = smem_size;

    hipLaunchAttribute attribute[1];
    attribute[0].id = cudaLaunchAttributeClusterDimension;
    attribute[0].val.clusterDim.x = ClusterSize;
    attribute[0].val.clusterDim.y = ClusterSize;
    attribute[0].val.clusterDim.z = 1;
    config.attrs = attribute;
    config.numAttrs = 1;

    cudaLaunchKernelEx(&config, matmul_gpu_cluster, d_input_a, d_input_b, d_output,
                       MatSize, Tile_size);

    hipMemcpy(gpu_output, d_output, MatSize * MatSize * sizeof(int),
               hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    for (int i = 0; i < MatSize; i++) {
        for (int j = 0; j < MatSize; j++) {
            int index = i * MatSize + j;
            EXPECT_EQ(output[index], gpu_output[index]);
        }
    }
}

